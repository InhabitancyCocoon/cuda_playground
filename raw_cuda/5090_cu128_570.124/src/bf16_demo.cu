
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_bf16.h>

// nvcc -o bf16_demo bf16_demo.cu 

int main(int argc, char* argv[]) {
    float fp32_x = 2.455;
    hip_bfloat16 bf16_x = fp32_x;
    std::cout << float(bf16_x) << "\n";  // 2.45312
    return 0;
}